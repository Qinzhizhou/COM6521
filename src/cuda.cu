#include "hip/hip_runtime.h"
#include "cuda.cuh"

#include <cstring>

#include "helper.h"

///
/// Algorithm storage
///
// Host copy of input image
Image cuda_input_image;
// Host copy of image tiles in each dimension
unsigned int cuda_TILES_X, cuda_TILES_Y;

// device copy of image's width and height
__device__ int d_width;
__device__ int d_height;
// device copy of nums of TILES in x-direction and y-direction
__device__ int d_TILES_X;
__device__ int d_TILES_Y;

//defien the function globally
__global__ void calMosaicSum(unsigned char* d_input_image_data, unsigned long long* d_mosaic_sum);

// Pointer to device buffer for calculating the sum of each tile mosaic, this must be passed to a kernel to be used on device
unsigned long long* d_mosaic_sum;
// Pointer to device buffer for storing the output pixels of each tile, this must be passed to a kernel to be used on device
unsigned char* d_mosaic_value;
// Pointer to device image data buffer, for storing the input image, this must be passed to a kernel to be used on device
unsigned char* d_input_image_data;
// Pointer to device image data buffer, for storing the output image data, this must be passed to a kernel to be used on device
unsigned char* d_output_image_data;
// Pointer to device buffer for the global pixel average sum, this must be passed to a kernel to be used on device
unsigned long long* d_global_pixel_sum;

void cuda_begin(const Image *input_image) {
    // These are suggested CUDA memory allocations that match the CPU implementation
    // If you would prefer, you can rewrite this function (and cuda_end()) to suit your preference
    cuda_TILES_X = input_image->width / TILE_SIZE;
    cuda_TILES_Y = input_image->height / TILE_SIZE;
    // Allocate buffer for calculating the sum of each tile mosaic
    CUDA_CALL(hipMalloc(&d_mosaic_sum, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned long long)));

    // Allocate buffer for storing the output pixel value of each tile
    CUDA_CALL(hipMalloc(&d_mosaic_value, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned char)));

    const size_t image_data_size = input_image->width * input_image->height * input_image->channels * sizeof(unsigned char);
    // Allocate copy of input image
    cuda_input_image = *input_image;
    cuda_input_image.data = (unsigned char*)malloc(image_data_size);
    memcpy(cuda_input_image.data, input_image->data, image_data_size);

    // zhou:  Allocate device copy of width and height
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_width),(void*)&input_image->width, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_height),(void*)&input_image->height, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_TILES_X),(void*)&cuda_TILES_X, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_TILES_Y),(void*)&cuda_TILES_Y, sizeof(int)));
    // Allocate and fill device buffer for storing input image data
    CUDA_CALL(hipMalloc(&d_input_image_data, image_data_size));
    CUDA_CALL(hipMemcpy(d_input_image_data, input_image->data, image_data_size, hipMemcpyHostToDevice));
    // Allocate device buffer for storing output image data
    CUDA_CALL(hipMalloc(&d_output_image_data, image_data_size));
    // Allocate and zero buffer for calculation global pixel average
    CUDA_CALL(hipMalloc(&d_global_pixel_sum, input_image->channels * sizeof(unsigned long long)));
}

// zhou: a global function for calculating mosaic sum

__global__ void calMosaicSum(unsigned char* d_input_image_data, unsigned long long*d_mosaic_sum){
    // use the indexes of threads to obtain tile_index and tile_offset. Here,
    // threadIdx.x is similar with t_x in cpu_stage1()
    // blockIdx.x is similar with t_y in cpu_stage1()
    // blockDim.x is similar with cpu_TILES_X in cpu_stage1()
    
    // Therefor, one thread in used for control one tile_index
    const unsigned int tile_index = (blockDim.x * blockIdx.x + threadIdx.x) * CHANNELS; // Tile_id
    const unsigned int tile_offset = (blockDim.x * blockIdx.x * TILE_SIZE * TILE_SIZE +  threadIdx.x * TILE_SIZE) * CHANNELS;// Offset
    
    // skip the threads which are out of index 
    if (tile_index >= d_TILES_X * d_TILES_Y * CHANNELS)
        return;
    
    // For each pixel within the tile
    for (int p_x = 0; p_x < TILE_SIZE; ++p_x) {
        for (int p_y = 0; p_y < TILE_SIZE; ++p_y) {
            // For each colour channel
            const unsigned int pixel_offset = (p_y * d_width + p_x) * CHANNELS;
            for (int ch = 0; ch < CHANNELS; ++ch) {
                // Load pixel
                const unsigned char pixel = d_input_image_data[tile_offset + pixel_offset + ch];
                d_mosaic_sum[tile_index + ch] += pixel;
            }
        }
    }
    return;
}

void cuda_stage1() { // 529.995 seconds in CPU 293.218 in GPU
    // initialize sizes of block and gird
    dim3 block = cuda_TILES_X; // 
    dim3 grid = ((cuda_TILES_X * cuda_TILES_Y + block.x - 1) / block.x);
    
    // Kernal function 
    calMosaicSum<<<block, grid>>>(d_input_image_data, d_mosaic_sum); // <<< 32, (32*32 + x - 1)/x 

#ifdef VALIDATION
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // delcare and allocate memory of test_mosaic_sum for validation
    unsigned long long* test_mosaic_sum = (unsigned long long*)malloc(cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long));
    
    // copy the data back to host
    CUDA_CALL(hipMemcpy(test_mosaic_sum, d_mosaic_sum, cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long), hipMemcpyDeviceToHost));
   validate_tile_sum(&cuda_input_image, test_mosaic_sum);
   #endif
}


void cuda_stage2(unsigned char* output_global_average) {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    // skip_compact_mosaic(cuda_TILES_X, cuda_TILES_Y, mosaic_sum, compact_mosaic, global_pixel_average);
    // Calculate the average of each tile, and sum these to produce a whole image average.
  


#ifdef VALIDATION
    // TODO: Uncomment and call the validation functions with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // validate_compact_mosaic(TILES_X, TILES_Y, mosaic_sum, mosaic_value, output_global_average);
#endif    
}






void cuda_stage3() {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    // skip_broadcast(cuda_input_image, compact_mosaic, d_output_image);

#ifdef VALIDATION
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // validate_broadcast(&input_image, mosaic_value, &output_image);
#endif    
}


void cuda_end(Image *output_image) {
    // This function matches the provided cuda_begin(), you may change it if desired

    // Store return value
    output_image->width = cuda_input_image.width;
    output_image->height = cuda_input_image.height;
    output_image->channels = cuda_input_image.channels;
    CUDA_CALL(hipMemcpy(output_image->data, d_output_image_data, output_image->width * output_image->height * output_image->channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
    // Release allocations
    free(cuda_input_image.data);
    CUDA_CALL(hipFree(d_mosaic_value));
    CUDA_CALL(hipFree(d_mosaic_sum));
    CUDA_CALL(hipFree(d_input_image_data));
    CUDA_CALL(hipFree(d_output_image_data));
}
