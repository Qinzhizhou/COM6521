#include "hip/hip_runtime.h"
#include "cuda.cuh"

#include <cstring>

#include "helper.h"

///
/// Algorithm storage
///
// Host copy of input image
Image cuda_input_image;
// Host copy of image tiles in each dimension
unsigned int cuda_TILES_X, cuda_TILES_Y;
// zhou: device copy of image's width and height
__device__ int d_width;
__device__ int d_height;
// zhou: device copy of nums of TILES in x-direction and y-direction
__device__ int d_TILES_X;
__device__ int d_TILES_Y;

// Pointer to device buffer for calculating the sum of each tile mosaic, this must be passed to a kernel to be used on device
unsigned long long* d_mosaic_sum;
// Pointer to device buffer for storing the output pixels of each tile, this must be passed to a kernel to be used on device
unsigned char* d_mosaic_value;
// Pointer to device image data buffer, for storing the input image, this must be passed to a kernel to be used on device
unsigned char* d_input_image_data;
// Pointer to device image data buffer, for storing the output image data, this must be passed to a kernel to be used on device
unsigned char* d_output_image_data;
// Pointer to device buffer for the global pixel average sum, this must be passed to a kernel to be used on device
unsigned long long* d_global_pixel_sum;

// global all the kernel function used in stage 1,2,3 
__global__ void calMosaicSum(unsigned char* d_input_image_data, unsigned long long* d_mosaic_sum);
__global__ void calMosaicValue(unsigned char* d_mosaic_value, unsigned long long* d_mosaic_sum);
__global__ void broadcastMosaicValueToPixel(unsigned char* d_mosaic_value, unsigned char* d_output_image_data);

void cuda_begin(const Image *input_image) {
    // These are suggested CUDA memory allocations that match the CPU implementation
    // If you would prefer, you can rewrite this function (and cuda_end()) to suit your preference
    cuda_TILES_X = input_image->width / TILE_SIZE;
    cuda_TILES_Y = input_image->height / TILE_SIZE;
    // Allocate buffer for calculating the sum of each tile mosaic
    CUDA_CALL(hipMalloc(&d_mosaic_sum, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned long long)));
    // Allocate buffer for storing the output pixel value of each tile
    CUDA_CALL(hipMalloc(&d_mosaic_value, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned char)));
    const size_t image_data_size = input_image->width * input_image->height * input_image->channels * sizeof(unsigned char);
    // Allocate copy of input image
    cuda_input_image = *input_image;
    cuda_input_image.data = (unsigned char*)malloc(image_data_size);
    memcpy(cuda_input_image.data, input_image->data, image_data_size);

    //Allocate device copy of width and height of image
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_width),(void*)&input_image->width, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_height),(void*)&input_image->height, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_TILES_X),(void*)&cuda_TILES_X, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_TILES_Y),(void*)&cuda_TILES_Y, sizeof(int)));

    // Allocate and fill device buffer for storing input image data
    CUDA_CALL(hipMalloc(&d_input_image_data, image_data_size));
    CUDA_CALL(hipMemcpy(d_input_image_data, input_image->data, image_data_size, hipMemcpyHostToDevice));

    // Allocate device buffer for storing output image data
    CUDA_CALL(hipMalloc(&d_output_image_data, image_data_size));

    // Allocate and zero buffer for calculation global pixel average
    CUDA_CALL(hipMalloc(&d_global_pixel_sum, input_image->channels * sizeof(unsigned long long)));
}

// zhou: a global function for calculating mosaic sum
__global__ void calMosaicSum(unsigned char* d_input_image_data, unsigned long long* d_mosaic_sum){
    // zhou: use the indexes of threads to obtain tile_index and tile_offset. Here,
    // threadIdx.x is similar with t_x in cpu_stage1()
    // blockIdx.x is similar with t_y in cpu_stage1()
    // blockDim.x is similar with cpu_TILES_X in cpu_stage1()
    // Therefor, one thread in used for control one tile_index
    const unsigned int tile_index = (blockDim.x * blockIdx.x + threadIdx.x) * CHANNELS;
    const unsigned int tile_offset = (blockDim.x * blockIdx.x * TILE_SIZE * TILE_SIZE +  threadIdx.x * TILE_SIZE) * CHANNELS;
    // skip the threads which are out of index 
    if (tile_index >= d_TILES_X * d_TILES_Y * CHANNELS)
        return;
    
    // For each pixel within the tile
    for (int p_x = 0; p_x < TILE_SIZE; ++p_x) {
        for (int p_y = 0; p_y < TILE_SIZE; ++p_y) {
            // For each colour channel
            const unsigned int pixel_offset = (p_y * d_width + p_x) * CHANNELS;
            for (int ch = 0; ch < CHANNELS; ++ch) {
                // Load pixel
                const unsigned char pixel = d_input_image_data[tile_offset + pixel_offset + ch];
                d_mosaic_sum[tile_index + ch] += pixel;
            }
        }
    }
    return;
}

void cuda_stage1() {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    // skip_tile_sum(input_image, mosaic_sum);

    // initialize sizes of block and gird, in particular block = 32 and gird = 32
    dim3 block = cuda_TILES_X;
    dim3 grid = ((cuda_TILES_X * cuda_TILES_Y + block.x - 1) / block.x);
    // call the global function calMosaicSum()
    calMosaicSum<<<grid, block>>>(d_input_image_data, d_mosaic_sum);

#ifdef VALIDATION
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    
    // zhou: delcare and allocate memory of test_mosaic_sum for validation
    unsigned long long* test_mosaic_sum = (unsigned long long*)malloc(cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long));
    // zhou:  copy the data back to host
    CUDA_CALL(hipMemcpy(test_mosaic_sum, d_mosaic_sum, cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long), hipMemcpyDeviceToHost));
   
   validate_tile_sum(&cuda_input_image, test_mosaic_sum);

   #endif
}

// a global function for calculating mosaic value, 
// which is equal to the corresponding mosaic sum over TILE_PIXELS in array d_mosaic_sum
// Since the global average is actually not used in picture mosaicization but only for validation
// we only calculate mosaic value for each tile in device but remain the calculation of global average in host.
__global__ void calMosaicValue(unsigned char* d_mosaic_value, unsigned long long* d_mosaic_sum){
    // use the indexes of threads to obtain tile_index_ch. Here,
    // (blockDim.x * blockIdx.x + threadIdx.x) is similar with the index (t * cpu_input_image.channels + ch) 
    // in cpu_stage2(). Therefor, one thread in used for control one mosaic value in a certain tile.
    const unsigned int tile_index_ch = (blockDim.x * blockIdx.x + threadIdx.x);
    // skip the threads which are out of index 
    if (tile_index_ch >= d_TILES_X * d_TILES_Y * CHANNELS)
        return;
    d_mosaic_value[tile_index_ch] = (unsigned char)(d_mosaic_sum[tile_index_ch] / TILE_PIXELS);  // Integer division is fine here
    return;
}
  
void cuda_stage2(unsigned char* output_global_average) {
    // Optionally during development call the skip function with the correct inputs to skip this stage
;

    // initialize sizes of block and gird, in particular block = 32 and gird = 32 * 3 = 96
    dim3 block = cuda_TILES_X; // 32
    dim3 grid = ((cuda_TILES_X * cuda_TILES_Y *  cuda_input_image.channels + block.x - 1) / block.x); //32 * 32 * 3
    // call the global function calMosaicValue()
    calMosaicValue<<<grid, block>>>(d_mosaic_value,  d_mosaic_sum);

    unsigned char* test_mosaic_value = (unsigned char*)malloc(cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned char));
    //copy the data back to host
    CUDA_CALL(hipMemcpy(test_mosaic_value, d_mosaic_value, cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Calculate sum of the average of each tile to produce a whole image average.
    unsigned long long whole_image_sum[4] = {0, 0, 0, 0};  // Only 3 is required for the assignment, but this version hypothetically supports upto 4 channels
    for (unsigned int t = 0; t < cuda_TILES_X * cuda_TILES_Y; ++t) {
        for (int ch = 0; ch < cuda_input_image.channels; ++ch) {
           whole_image_sum[ch] += test_mosaic_value[t * cuda_input_image.channels + ch];
        }
    }
    for (int ch = 0; ch < CHANNELS; ++ch) {
        output_global_average[ch] = (unsigned char)(whole_image_sum[ch] / (cuda_TILES_X * cuda_TILES_Y));
    }
    
#ifdef VALIDATION
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // delcare and allocate memory of test_mosaic_value for validation
    // delcare and allocate memory of test_mosaic_sum for validation

    unsigned long long* test_mosaic_sum = (unsigned long long*)malloc(cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long));
    // zhou:  copy the data back to host
    CUDA_CALL(hipMemcpy(test_mosaic_sum, d_mosaic_sum, cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long), hipMemcpyDeviceToHost));
    validate_compact_mosaic(cuda_TILES_X, cuda_TILES_Y, test_mosaic_sum, test_mosaic_value, output_global_average);

#endif    
}

// za global function for broadcasting average mosaic value to pixel in a certain tile
__global__ void broadcastMosaicValueToPixel(unsigned char* d_mosaic_value, unsigned char* d_output_image_data){

    // i and j are global thread (x- and y-direction respectively) indexes in total grids
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
    // declare static share memory with length 3 
    __shared__ unsigned char shareMemory[CHANNELS];
    
    //  blockIdx and gridDim are used to obtain tile_index
    const unsigned int tile_index = blockIdx.y *  gridDim.x + blockIdx.x;

    // only the first thread in each block is used to initialize share momery
    if  ( threadIdx.x == 0 &&  threadIdx.y == 0 ){
        for (int ch = 0; ch < CHANNELS; ++ch) {
            shareMemory[ch] = d_mosaic_value[tile_index * CHANNELS + ch];
        }
    }
    // sync all the threads in a certain block, 
    // since all of them can not continue until the the shared memory loaded.
    __syncthreads();

    //  thread global index i, j and gridDim.x and blockDim.x are used to obtain pixel_global_index 
    // in particular gridDim.x * blockDim.x = 128 * 32 = 4096
    const unsigned int pixel_global_index = (j * gridDim.x * blockDim.x + i) * CHANNELS;

    if (pixel_global_index >= d_TILES_X * d_TILES_Y * TILE_PIXELS * CHANNELS)
        return;

    // zhou: broadcast mosaic value from share memory to each pixel
    // every thread control 3 channels for each pixel
    // obtaining corrsponding mosaic value from share memory
    for (int ch = 0; ch < CHANNELS; ++ch) {
        d_output_image_data[pixel_global_index + ch] = shareMemory[ch];
    }
    return;
}

void cuda_stage3() { // broadcast function    
    // initialize sizes of block and gird, using 2D block and grid here
    // in particular block = (32, 32) and grid = (128, 128)
    dim3 block = dim3(TILE_SIZE, TILE_SIZE);
    dim3 grid = dim3((cuda_input_image.width + block.x - 1) / block.x,  (cuda_input_image.height + block.y - 1) / block.y );
    // call the global function broadcastMosaicValueToPixel()
    broadcastMosaicValueToPixel<<<grid, block>>>(d_mosaic_value, d_output_image_data);

#ifdef VALIDATION
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)

    // delcare and allocate memory of test_mosaic_value for validation
    unsigned char* test_mosaic_value = (unsigned char*)malloc(cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned char));
    // copy the data back to host
    CUDA_CALL(hipMemcpy(test_mosaic_value, d_mosaic_value, cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // calculate the size of image data, in particular 4096*4096*3 //
    const size_t image_data_size = cuda_input_image.width * cuda_input_image.height * cuda_input_image.channels * sizeof(unsigned char);
    // delcare and allocate memory of test_output_image for validation
    Image test_output_image = cuda_input_image;
    test_output_image.data = (unsigned char*)malloc(image_data_size);
    // copy the data back to host
    CUDA_CALL(hipMemcpy(test_output_image.data, d_output_image_data, image_data_size, hipMemcpyDeviceToHost));

    validate_broadcast(&cuda_input_image, test_mosaic_value, &test_output_image);

#endif    
}

void cuda_end(Image *output_image) {
    // This function matches the provided cuda_begin(), you may change it if desired

    // Store return value
    output_image->width = cuda_input_image.width;
    output_image->height = cuda_input_image.height;
    output_image->channels = cuda_input_image.channels;
    CUDA_CALL(hipMemcpy(output_image->data, d_output_image_data, output_image->width * output_image->height * output_image->channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
    // Release allocations
    free(cuda_input_image.data);
    CUDA_CALL(hipFree(d_mosaic_value));
    CUDA_CALL(hipFree(d_mosaic_sum));
    CUDA_CALL(hipFree(d_input_image_data));
    CUDA_CALL(hipFree(d_output_image_data));
    CUDA_CALL(hipFree(d_global_pixel_sum));
}
